
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA 核函数进行矩阵乘法
__global__ void matrixMultiply(double *A, double *B, double *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // 当前处理的行
    int col = blockIdx.x * blockDim.x + threadIdx.x; // 当前处理的列

    if (row < m && col < n) {
        double sum = 0.0;
        for (int p = 0; p < k; p++) {
            sum += A[row * k + p] * B[p * n + col];
        }
        C[row * n + col] = sum;
    }
}

// 主机函数，调用 CUDA 核函数
extern "C" void MY_MMult(int m, int n, int k, double *a, int lda,
                         double *b, int ldb,
                         double *c, int ldc) {
    double *d_A, *d_B, *d_C;

    size_t sizeA = m * k * sizeof(double);
    size_t sizeB = k * n * sizeof(double);
    size_t sizeC = m * n * sizeof(double);

    // 在设备上分配内存
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    // 将 A 和 B 从主机复制到设备
    hipMemcpy(d_A, a, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, sizeB, hipMemcpyHostToDevice);

    // 定义线程块和网格的维度
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用 CUDA 核函数
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);

    // 将结果从设备复制回主机
    hipMemcpy(c, d_C, sizeC, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
